#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/ds_triplet_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/io.hpp" 

namespace caffe {

template <typename Dtype>
void DsTripletLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  caffe_gpu_sub(  
      count,  
      bottom[2]->gpu_data(),  // f(x_i^n)
      bottom[1]->gpu_data(),  // f(x_i^p)
      diff_np_.mutable_gpu_data());  // f(x_i^n)-f(x_i^p)
  caffe_gpu_sub(  
      count,  
      bottom[0]->gpu_data(),  // f(x_i^a)
      bottom[1]->gpu_data(),  // f(x_i^p)
      diff_ap_.mutable_gpu_data());  // f(x_i^a)-f(x_i^p)
  caffe_gpu_sub(  
      count,  
      bottom[0]->gpu_data(),  // f(x_i^a)
      bottom[2]->gpu_data(),  // f(x_i^n)
      diff_an_.mutable_gpu_data());  // f(x_i^a)-f(x_i^n)
  caffe_gpu_powx(  
      count,  
      diff_ap_.mutable_gpu_data(),  // f(x_i^a)-f(x_i^p)
      Dtype(2),  
      diff_ap_sq_.mutable_gpu_data());  // (f(x_i^a)-f(x_i^p)).^2 
  caffe_gpu_powx(  
      count,  
      diff_an_.mutable_gpu_data(),  // f(x_i^a)-f(x_i^n)
      Dtype(2),  
      diff_an_sq_.mutable_gpu_data());  // (f(x_i^a)-f(x_i^n)).^2 
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),               //alpha
      diff_ap_sq_.gpu_data(),   // A :  (f(x_i^a)-f(x_i^p)).^2 
      summer_vec_.gpu_data(),   // x :  [ 1, 1, 1,...,1 ]
      Dtype(0.0),               // beta
      dist_ap_sq_.mutable_gpu_data());  // y :||f(x_i^a)-f(x_i^p)||^2 
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),               //alpha 
      diff_an_sq_.gpu_data(),   // A :  (f(x_i^a)-f(x_i^n)).^2 
      summer_vec_.gpu_data(),   // x :  [ 1, 1, 1,...,1 ]
      Dtype(0.0),               // beta
      dist_an_sq_.mutable_gpu_data());  // y :||f(x_i^a)-f(x_i^n)||^2 
  Dtype margin = this->layer_param_.ds_triplet_loss_param().margin();
  Dtype loss(0.0);
for (int i = 0; i < bottom[0]->num(); ++i) {  
     loss += std::max(margin +dist_ap_sq_.cpu_data()[i]- dist_an_sq_.cpu_data()[i], Dtype(0.0));  
  }  
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>  
__global__ void CLLBackward(const int count, const int channels,  
    const Dtype margin, const Dtype alpha, const Dtype* diff,
    const Dtype* dist_ap_sq, const Dtype* dist_an_sq,  
    Dtype *bottom_diff) {  
  CUDA_KERNEL_LOOP(i, count) {  
    int n = i / channels;  // the num index, to access dist_ap_sq_ and dist_an_sq_  
    Dtype trip_dist(0.0);  
    trip_dist= margin + dist_ap_sq[n] - dist_an_sq[n];  
    if (trip_dist> 0.0) {  
        bottom_diff[i] = alpha * diff[i];  
    } else {  
        bottom_diff[i] = 0;  
    }  
  }  
}  

template <typename Dtype>
void DsTripletLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  Dtype margin = this->layer_param_.ds_triplet_loss_param().margin(); 
  const int count = bottom[0]->count();  
  const int channels = bottom[0]->channels(); 

  for (int i = 0; i < 3; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 1) ? -2 : 2;
      const Dtype alpha = sign * top[0]->cpu_diff()[0];
      // NOLINT_NEXT_LINE(whitespace/operators)
      if(i==0){     // \frac{\partial(L)}{\partial f(x_i^a)}
          // NOLINT_NEXT_LINE(whitespace/operators)  
          CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(  
              count, channels, margin, alpha,  
              diff_np_.gpu_data(),  //  f(x_i^n)-f(x_i^p)
              dist_ap_sq_.gpu_data(),  // ||f(x_i^a)-f(x_i^p)||^2   
              dist_an_sq_.gpu_data(),  // ||f(x_i^a)-f(x_i^n)||^2 
              bottom[i]->mutable_gpu_diff());  
          CUDA_POST_KERNEL_CHECK;  
      }else if(i==1){    // \frac{\partial(L)}{\partial f(x_i^p)}
          // NOLINT_NEXT_LINE(whitespace/operators)  
          CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(  
              count, channels, margin, alpha,   
              diff_ap_.gpu_data(),  //  f(x_i^a)-f(x_i^p)
              dist_ap_sq_.gpu_data(),  // ||f(x_i^a)-f(x_i^p)||^2   
              dist_an_sq_.gpu_data(),  // ||f(x_i^a)-f(x_i^n)||^2   
              bottom[i]->mutable_gpu_diff());  
          CUDA_POST_KERNEL_CHECK;  
      }else if(i==2){   // \frac{\partial(L)}{\partial f(x_i^n)}
          // NOLINT_NEXT_LINE(whitespace/operators)  
          CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(  
              count, channels, margin, alpha,  
              diff_an_.gpu_data(),  //  f(x_i^a)-f(x_i^n)
              dist_ap_sq_.gpu_data(),  // ||f(x_i^a)-f(x_i^p)||^2   
              dist_an_sq_.gpu_data(),  // ||f(x_i^a)-f(x_i^n)||^2 
              bottom[i]->mutable_gpu_diff());  
          CUDA_POST_KERNEL_CHECK;  
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DsTripletLossLayer);

}  // namespace caffe